#include "hip/hip_runtime.h"
#include <windows.h>
#include "avisynth.h"

#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

int nblocks(int n, int block) {
	return (n + block - 1) / block;
}

class InvertNeg : public GenericVideoFilter {
public:
	InvertNeg(PClip _child, IScriptEnvironment* env);
	PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env);
  int __stdcall SetCacheHints(int cachehints, int frame_range);
};

InvertNeg::InvertNeg(PClip _child, IScriptEnvironment* env) :
GenericVideoFilter(_child) {
	if (!vi.IsPlanar() || !vi.IsYUV()) {
		env->ThrowError("InvertNeg: planar YUV data only!");
	}
}

__global__ void InvertNegKernel(
	const unsigned char* srcp, unsigned char* dstp,
	int src_pitch, int dst_pitch, int row_size, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < row_size && y < height) {
		dstp[x + y * dst_pitch] = srcp[x + y * src_pitch] ^ 255;
	}
}

PVideoFrame __stdcall InvertNeg::GetFrame(int n, IScriptEnvironment* env_)
{
  IScriptEnvironment2* env = static_cast<IScriptEnvironment2*>(env_);

  if (env->GetProperty(AEP_DEVICE_TYPE) != DEV_TYPE_CUDA) {
    env->ThrowError("InvertNeg: Only CUDA frame is supported.");
  }

	PVideoFrame src = child->GetFrame(n, env);
	PVideoFrame dst = env->NewVideoFrame(vi);

	int planes[] = { PLANAR_Y, PLANAR_V, PLANAR_U };

	for (int p = 0; p<3; p++) {
    const unsigned char* srcp = src->GetReadPtr(planes[p]);
    unsigned char* dstp = dst->GetWritePtr(planes[p]);

    int src_pitch = src->GetPitch(planes[p]);
    int dst_pitch = dst->GetPitch(planes[p]);
    int row_size = dst->GetRowSize(planes[p]);
    int height = dst->GetHeight(planes[p]);

		dim3 threads(32, 16);
		dim3 blocks(nblocks(row_size, threads.x), nblocks(height, threads.y));
		InvertNegKernel << <blocks, threads >> >(srcp, dstp, src_pitch, dst_pitch, row_size, height);
	}
	return dst;
}

int __stdcall InvertNeg::SetCacheHints(int cachehints, int frame_range)
{
  if (cachehints == CACHE_GET_MTMODE)
    return MT_NICE_FILTER;
  if (cachehints == CACHE_GET_DEV_TYPE)
    return DEV_TYPE_CUDA; // Only CUDA is supported

  return 0;
}

AVSValue __cdecl Create_InvertNeg(AVSValue args, void* user_data, IScriptEnvironment* env) {
	return new InvertNeg(args[0].AsClip(), env);
}

const AVS_Linkage *AVS_linkage = 0;

extern "C" __declspec(dllexport) const char* __stdcall AvisynthPluginInit3(IScriptEnvironment* env, const AVS_Linkage* const vectors) {
	AVS_linkage = vectors;
	env->AddFunction("InvertNeg", "c", Create_InvertNeg, 0);
	return "CUDA InvertNeg sample plugin";
}
